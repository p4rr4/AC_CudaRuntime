﻿/*----------------------------------------------------------------------------*/
/*  FICHERO:       calculaNormales.cu									          */
/*  AUTOR:         Jorge Azorin											  */
/*													                          */
/*  RESUMEN												                      */
/*  ~~~~~~~												                      */
/* Ejercicio grupal para el cálculo de las normales de una superficie          */
/*----------------------------------------------------------------------------*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>


// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "calculaNormales.cuh"
#include <Windows.h>



#define ERROR_CHECK { hipError_t err; if ((err = hipGetLastError()) != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

typedef LARGE_INTEGER timeStamp;
double getTime();

/*----------------------------------------------------------------------------*/
/*  FUNCION A PARALELIZAR  (versión secuencial-CPU)  				          */
/*	Cálculo de las normales de una superficie definida por una                */
/*  una malla de vtotal x utotal puntos 3D                                    */
/*----------------------------------------------------------------------------*/
int CalculoNormalesCPU()
{
	TPoint3D direct1, direct2, normal;
	int vecindadU[9] = { -1,0,1,1,1,0,-1,-1,-1 }; // Vecindad 8 + 1 para calcular todas las rectas
	int vecindadV[9] = { -1,-1,-1,0,1,1,1,0,-1 };
	int vV, vU;
	int numDir;
	int oKdir1, oKdir2;
	/* La vecindad es:
	*--*--*
	|  |  |
	*--X--*
	|  |  |
	*--*--*
	*/
	int cont = 0;

	for (int u = 0; u < S.UPoints; u++)			// Recorrido de todos los puntos de la superficie
	{
		for (int v = 0; v < S.VPoints; v++)
		{
			normal.x = 0;
			normal.y = 0;
			normal.z = 0;
			numDir = 0;
			for (int nv = 0; nv < 8; nv++)  // Para los puntos de la vecindad
			{
				vV = v + vecindadV[nv];
				vU = u + vecindadU[nv];
				if (vV >= 0 && vU >= 0 && vV < S.VPoints && vU < S.UPoints)
				{
					direct1.x = S.Buffer[v][u].x - S.Buffer[vV][vU].x;
					direct1.y = S.Buffer[v][u].y - S.Buffer[vV][vU].y;
					direct1.z = S.Buffer[v][u].z - S.Buffer[vV][vU].z;
					oKdir1 = 1;
				}
				else
				{
					direct1.x = 0.0;
					direct1.y = 0.0;
					direct1.z = 0.0;
					oKdir1 = 0;
				}
				vV = v + vecindadV[nv + 1];
				vU = v + vecindadU[nv + 1];

				if (vV >= 0 && vU >= 0 && vV < S.VPoints && vU < S.UPoints)
				{
					direct2.x = S.Buffer[v][u].x - S.Buffer[vV][vU].x;
					direct2.y = S.Buffer[v][u].y - S.Buffer[vV][vU].y;
					direct2.z = S.Buffer[v][u].z - S.Buffer[vV][vU].z;
					oKdir2 = 1;
				}
				else
				{
					direct2.x = 0.0;
					direct2.y = 0.0;
					direct2.z = 0.0;
					oKdir2 = 0;
				}
				if (oKdir1 == 1 && oKdir2 == 1)
				{
					normal.x += direct1.y * direct2.z - direct1.z * direct2.y;
					normal.y += direct1.x * direct2.z - direct1.z * direct2.x;
					normal.z += direct1.x * direct2.y - direct1.y * direct2.x;
					numDir++;
				}
			}
			NormalUCPU[cont] = normal.x / (float)numDir;
			NormalVCPU[cont] = normal.y / (float)numDir;
			NormalWCPU[cont] = normal.z / (float)numDir;
			cont++;
		}
	}

	return OKCALC;									// Simulación CORRECTA
}

// ---------------------------------------------------------------
// ---------------------------------------------------------------
// FUNCION A IMPLEMENTAR POR EL GRUPO (paralelización de CalculoNormalesCPU)
// ---------------------------------------------------------------
// ---------------------------------------------------------------

int CalculoNormalesGPU()
{
	return OKCALC;
}
// ---------------------------------------------------------------
// ---------------------------------------------------------------
// ---------------------------------------------------------------
// ---------------------------------------------------------------
// ---------------------------------------------------------------

// Declaraciones adelantadas de funciones
int LeerSuperficie(const char* fichero);



////////////////////////////////////////////////////////////////////////////////
//PROGRAMA PRINCIPAL
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{


	double gpu_start_time, gpu_end_time;
	double cpu_start_time, cpu_end_time;

	/* Numero de argumentos */
	if (argc != 2)
	{
		fprintf(stderr, "Numero de parametros incorecto\n");
		fprintf(stderr, "Uso: %s superficie\n", argv[0]);
		return;
	}

	/* Apertura de Fichero */
	printf("Cálculo de las normales de la superficie...\n");
	/* Datos de la superficie */
	if (LeerSuperficie((char*)argv[1]) == ERRORCALC)
	{
		fprintf(stderr, "Lectura de superficie incorrecta\n");
		return;
	}
	int numPuntos;
	numPuntos = S.UPoints * S.VPoints;

	// Creación buffer resultados para versiones CPU y GPU
	NormalVCPU = (float*)malloc(numPuntos * sizeof(float));
	NormalUCPU = (float*)malloc(numPuntos * sizeof(float));
	NormalWCPU = (float*)malloc(numPuntos * sizeof(float));
	NormalVGPU = (float*)malloc(numPuntos * sizeof(float));
	NormalUGPU = (float*)malloc(numPuntos * sizeof(float));
	NormalWGPU = (float*)malloc(numPuntos * sizeof(float));

	/* Algoritmo a paralelizar */
	cpu_start_time = getTime();
	if (CalculoNormalesCPU() == ERRORCALC)
	{
		fprintf(stderr, "Cálculo CPU incorrecta\n");
		BorrarSuperficie();
		if (NormalVCPU != NULL) free(NormalVCPU);
		if (NormalUCPU != NULL) free(NormalUCPU);
		if (NormalWCPU != NULL) free(NormalUCPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		if (NormalWGPU != NULL) free(NormalVGPU);
		if (NormalUGPU != NULL) free(NormalUGPU);		exit(1);
	}
	cpu_end_time = getTime();
	/* Algoritmo a implementar */
	gpu_start_time = getTime();
	if (CalculoNormalesGPU() == ERRORCALC)
	{
		fprintf(stderr, "Cálculo GPU incorrecta\n");
		BorrarSuperficie();
		if (NormalVCPU != NULL) free(NormalVCPU);
		if (NormalUCPU != NULL) free(NormalUCPU);
		if (NormalWCPU != NULL) free(NormalUCPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		if (NormalUGPU != NULL) free(NormalUGPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		return;
	}
	gpu_end_time = getTime();
	// Comparación de corrección
	int comprobar = OKCALC;
	for (int i = 0; i < numPuntos; i++)
	{
		if (((int)NormalVCPU[i] * 1000 != (int)NormalVGPU[i]) * 1000 || ((int)NormalUCPU[i] * 1000 != (int)NormalUGPU[i] * 1000) || ((int)NormalWCPU[i] * 1000 != (int)NormalWGPU[i] * 1000))
		{
			comprobar = ERRORCALC;
			fprintf(stderr, "Fallo en el punto %d, valor correcto V=%f U=%f W=%f\n", i, NormalVCPU[i], NormalUCPU[i], NormalWCPU[i]);
		}
	}
	// Impresion de resultados
	if (comprobar == OKCALC)
	{
		printf("Cálculo correcto!\n");

	}
	// Impresión de resultados
	printf("Tiempo ejecución GPU : %fs\n", \
		gpu_end_time - gpu_start_time);
	printf("Tiempo de ejecución en la CPU : %fs\n", \
		cpu_end_time - cpu_start_time);
	printf("Se ha conseguido un factor de aceleración %fx utilizando CUDA\n", (cpu_end_time - cpu_start_time) / (gpu_end_time - gpu_start_time));
	// Limpieza de buffers
	BorrarSuperficie();
	if (NormalVCPU != NULL) free(NormalVCPU);
	if (NormalUCPU != NULL) free(NormalUCPU);
	if (NormalWCPU != NULL) free(NormalWCPU);
	if (NormalVGPU != NULL) free(NormalVGPU);
	if (NormalUGPU != NULL) free(NormalUGPU);
	if (NormalWGPU != NULL) free(NormalWGPU);
	return;
}

int
main(int argc, char** argv)
{
	runTest(argc, argv);
	getchar();
}

/* Funciones auxiliares */
double getTime()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}



/*----------------------------------------------------------------------------*/
/*	Función:  LeerSuperficie(char *fichero)						              */
/*													                          */
/*	          Lee los datos de la superficie de un fichero con formato .FOR   */
/*----------------------------------------------------------------------------*/
int LeerSuperficie(const char* fichero)
{
	int i, j, count;		/* Variables de bucle */
	int utotal, vtotal;		/* Variables de tamaño de superficie */
	FILE* fpin; 			/* Fichero */
	double x, y, z;

	/* Apertura de Fichero */
	if ((fpin = fopen(fichero, "r")) == NULL) return ERRORCALC;
	/* Lectura de cabecera */
	if (fscanf(fpin, "Ancho=%d\n", &utotal) < 0) return ERRORCALC;
	if (fscanf(fpin, "Alto=%d\n", &vtotal) < 0) return ERRORCALC;
	if (utotal * vtotal <= 0) return ERRORCALC;
	/* Localizacion de comienzo */
	if (feof(fpin)) return ERRORCALC;
	/* Inicialización de parametros geometricos */
	if (CrearSuperficie(utotal, vtotal) == ERRORCALC) return ERRORCALC;
	/* Lectura de coordenadas */
	count = 0;
	for (i = 0; i < utotal; i++)
	{
		for (j = 0; j < vtotal; j++)
		{
			if (!feof(fpin))
			{
				fscanf(fpin, "%lf %lf %lf\n", &x, &y, &z);
				S.Buffer[j][i].x = x;
				S.Buffer[j][i].y = y;
				S.Buffer[j][i].z = z;
				count++;
			}
			else break;
		}
	}
	fclose(fpin);
	if (count != utotal * vtotal) return ERRORCALC;
	return OKCALC;
}



