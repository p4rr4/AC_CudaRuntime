#include "hip/hip_runtime.h"
﻿/*----------------------------------------------------------------------------*/
/*  FICHERO:       calculaNormales.cu									          */
/*  AUTOR:         Jorge Azorin											  */
/*													                          */
/*  RESUMEN												                      */
/*  ~~~~~~~												                      */
/* Ejercicio grupal para el cálculo de las normales de una superficie          */
/*----------------------------------------------------------------------------*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>


// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "calculaNormales.cuh"
#include <Windows.h>



#define ERROR_CHECK { hipError_t err; if ((err = hipGetLastError()) != hipSuccess) { printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

typedef LARGE_INTEGER timeStamp;
double getTime();

/*----------------------------------------------------------------------------*/
/*  FUNCION A PARALELIZAR  (versión secuencial-CPU)  				          */
/*	Cálculo de las normales de una superficie definida por una                */
/*  una malla de vtotal x utotal puntos 3D                                    */
/*----------------------------------------------------------------------------*/
int CalculoNormalesCPU()
{
	TPoint3D direct1, direct2, normal;
	int vecindadU[9] = { -1,0,1,1,1,0,-1,-1,-1 }; // Vecindad 8 + 1 para calcular todas las rectas
	int vecindadV[9] = { -1,-1,-1,0,1,1,1,0,-1 };
	int vV, vU;
	int numDir;
	int oKdir1, oKdir2;
	/* La vencidad es:
	*--*--*
	|  |  |
	*--X--*
	|  |  |
	*--*--*
	*/
	int cont = 0;

	for (int u = 0; u < S.UPoints; u++)			// Recorrido de todos los puntos de la superficie
	{
		for (int v = 0; v < S.VPoints; v++)
		{
			normal.x = 0;
			normal.y = 0;
			normal.z = 0;
			numDir = 0;
			for (int nv = 0; nv < 8; nv++)  // Para los puntos de la vecindad
			{
				vV = v + vecindadV[nv];
				vU = u + vecindadU[nv];
				if (vV >= 0 && vU >= 0 && vV < S.VPoints && vU < S.UPoints)
				{
					direct1.x = S.Buffer[v][u].x - S.Buffer[vV][vU].x;
					direct1.y = S.Buffer[v][u].y - S.Buffer[vV][vU].y;
					direct1.z = S.Buffer[v][u].z - S.Buffer[vV][vU].z;
					oKdir1 = 1;
				}
				else
				{
					direct1.x = 0.0;
					direct1.y = 0.0;
					direct1.z = 0.0;
					oKdir1 = 0;
				}
				vV = v + vecindadV[nv + 1];
				vU = v + vecindadU[nv + 1];

				if (vV >= 0 && vU >= 0 && vV < S.VPoints && vU < S.UPoints)
				{
					direct2.x = S.Buffer[v][u].x - S.Buffer[vV][vU].x;
					direct2.y = S.Buffer[v][u].y - S.Buffer[vV][vU].y;
					direct2.z = S.Buffer[v][u].z - S.Buffer[vV][vU].z;
					oKdir2 = 1;
				}
				else
				{
					direct2.x = 0.0;
					direct2.y = 0.0;
					direct2.z = 0.0;
					oKdir2 = 0;
				}
				if (oKdir1 == 1 && oKdir2 == 1)
				{
					normal.x += direct1.y * direct2.z - direct1.z * direct2.y;
					normal.y += direct1.x * direct2.z - direct1.z * direct2.x;
					normal.z += direct1.x * direct2.y - direct1.y * direct2.x;
					numDir++;
				}
			}
			NormalUCPU[cont] = normal.x / (float)numDir;
			NormalVCPU[cont] = normal.y / (float)numDir;
			NormalWCPU[cont] = normal.z / (float)numDir;
			cont++;
		}
	}

	return OKCALC;								// Simulación CORRECTA
}

// ---------------------------------------------------------------
// ---------------------------------------------------------------
// FUNCION A IMPLEMENTAR POR EL GRUPO (paralelización de CalculoNormalesCPU)
// ---------------------------------------------------------------
// ---------------------------------------------------------------

__global__ void CalculoNormalesGPU(TPoint3D* Buffer, float* NormalV, float* NormalU, float* NormalW, int UPoints, int VPoints)
{
	int u = blockIdx.x * blockDim.x + threadIdx.x;
    int v = blockIdx.y * blockDim.y + threadIdx.y;
	int index = v * UPoints + u;

	if (u < UPoints && v < VPoints)
	{
		TPoint3D direct1, direct2, normal;
		int vecindadU[9] = { -1,0,1,1,1,0,-1,-1,-1 }; // Vecindad 8 + 1 para calcular todas las rectas
		int vecindadV[9] = { -1,-1,-1,0,1,1,1,0,-1 };
		int vV, vU;
		int numDir;
		int oKdir1, oKdir2;

		normal.x = 0;
		normal.y = 0;
		normal.z = 0;
		numDir = 0;

		for (int nv = 0; nv < 8; nv++)  // Para los puntos de la vecindad
		{
			vV = v + vecindadV[nv];
			vU = u + vecindadU[nv];

			if (vV >= 0 && vU >= 0 && vV < VPoints && vU < UPoints) {
				direct1.x = Buffer[v * UPoints + u].x - Buffer[vV * UPoints + vU].x;
				direct1.y = Buffer[v * UPoints + u].y - Buffer[vV * UPoints + vU].y;
				direct1.z = Buffer[v * UPoints + u].z - Buffer[vV * UPoints + vU].z;
				oKdir1 = 1;
			}
			else {
				direct1.x = direct1.y = direct1.z = 0.0;
			}

			vV = v + vecindadV[nv + 1];
			vU = u + vecindadU[nv + 1];

			if (vV >= 0 && vU >= 0 && vV < VPoints && vU < UPoints) {
				direct2.x = Buffer[v * UPoints + u].x - Buffer[vV * UPoints + vU].x;
				direct2.y = Buffer[v * UPoints + u].y - Buffer[vV * UPoints + vU].y;
				direct2.z = Buffer[v * UPoints + u].z - Buffer[vV * UPoints + vU].z;
				oKdir2 = 1;
			}
			else {
				direct2.x = direct2.y = direct2.z = 0.0;
			}

			if (oKdir1 == 1 && oKdir2 == 1) {
				normal.x += direct1.y * direct2.z - direct1.z * direct2.y;
				normal.y += direct1.x * direct2.z - direct1.z * direct2.x;
				normal.z += direct1.x * direct2.y - direct1.y * direct2.x;
				numDir++;
			}
		}
		NormalU[index] = normal.x / (float)numDir;
		NormalV[index] = normal.y / (float)numDir;
		NormalW[index] = normal.z / (float)numDir;
	}
}

// ---------------------------------------------------------------
// ---------------------------------------------------------------
// ---------------------------------------------------------------
// ---------------------------------------------------------------
// ---------------------------------------------------------------

// Declaraciones adelantadas de funciones
int LeerSuperficie(const char* fichero);



////////////////////////////////////////////////////////////////////////////////
//PROGRAMA PRINCIPAL
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{
	double gpu_start_time, gpu_end_time;
	double cpu_start_time, cpu_end_time;

	/* Numero de argumentos */
	if (argc != 2)
	{
		fprintf(stderr, "Numero de parametros incorecto\n");
		fprintf(stderr, "Uso: %s superficie\n", argv[0]);
		return;
	}

	/* Apertura de Fichero */
	printf("Calculo de las normales de la superficie...\n");
	/* Datos de la superficie */
	if (LeerSuperficie((char*)argv[1]) == ERRORCALC)
	{
		fprintf(stderr, "Lectura de superficie incorrecta\n");
		return;
	}
	int numPuntos;
	numPuntos = S.UPoints * S.VPoints;

	// GPU Surface
	TPoint3D* Buffer_d;

	hipMalloc(&Buffer_d, numPuntos * sizeof(TPoint3D));
	hipMemcpy(Buffer_d, S.Buffer, numPuntos * sizeof(TPoint3D), hipMemcpyHostToDevice);

	// Creación buffer resultados para versiones CPU
	NormalVCPU = (float*)malloc(numPuntos * sizeof(float));
	NormalUCPU = (float*)malloc(numPuntos * sizeof(float));
	NormalWCPU = (float*)malloc(numPuntos * sizeof(float));

	// Creación buffer resultados para versiones GPU
	NormalVGPU = (float*)malloc(numPuntos * sizeof(float));
	NormalUGPU = (float*)malloc(numPuntos * sizeof(float));
	NormalWGPU = (float*)malloc(numPuntos * sizeof(float));

	// Results buffers in GPU device
	float* NormalVGPU_d;
	float* NormalUGPU_d;
	float* NormalWGPU_d;

	hipMalloc(&NormalVGPU_d, numPuntos * sizeof(float));
	hipMalloc(&NormalUGPU_d, numPuntos * sizeof(float));
	hipMalloc(&NormalWGPU_d, numPuntos * sizeof(float));

	/* Algoritmo a paralelizar */
	cpu_start_time = getTime();
	if (CalculoNormalesCPU() == ERRORCALC)
	{
		fprintf(stderr, "Calculo CPU incorrecta\n");
		BorrarSuperficie();
		if (NormalVCPU != NULL) free(NormalVCPU);
		if (NormalUCPU != NULL) free(NormalUCPU);
		if (NormalWCPU != NULL) free(NormalUCPU);
		if (NormalVGPU != NULL) free(NormalVGPU);
		if (NormalWGPU != NULL) free(NormalVGPU);
		if (NormalUGPU != NULL) free(NormalUGPU);		exit(1);
	}
	cpu_end_time = getTime();

	// Define grid and block dimensions
	dim3 blockSize(16, 16);
	dim3 gridSize((S.UPoints + blockSize.x - 1) / blockSize.x, (S.VPoints + blockSize.y - 1) / blockSize.y);

	gpu_start_time = getTime();

	// Start the kernel
	CalculoNormalesGPU<<<gridSize, blockSize>>>(Buffer_d, NormalVGPU_d, NormalUGPU_d, NormalWGPU_d, S.UPoints, S.VPoints);
	ERROR_CHECK;  // Comprobar errores después de lanzar el kernel
	hipDeviceSynchronize();
	ERROR_CHECK;

	hipDeviceSynchronize();

	gpu_end_time = getTime();

	hipMemcpy(NormalVGPU, NormalVGPU_d, numPuntos * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(NormalUGPU, NormalUGPU_d, numPuntos * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(NormalWGPU, NormalWGPU_d, numPuntos * sizeof(float), hipMemcpyDeviceToHost);

	// Comparación de corrección
	int comprobar = OKCALC;
	for (int i = 0; i < numPuntos; i++)
	{
		printf("%f %f %f\n", NormalVCPU[i], NormalUCPU[i], NormalWCPU[i]);
		printf("%f %f %f\n", NormalVGPU[i], NormalUGPU[i], NormalWGPU[i]);

		/*if (((int)NormalVCPU[i] * 1000 != (int)NormalVGPU[i]) * 1000 || ((int)NormalUCPU[i] * 1000 != (int)NormalUGPU[i] * 1000) || ((int)NormalWCPU[i] * 1000 != (int)NormalWGPU[i] * 1000))
		{
			comprobar = ERRORCALC;
			fprintf(stderr, "Fallo en el punto %d, valor correcto V=%f U=%f W=%f\n", i, NormalVCPU[i], NormalUCPU[i], NormalWCPU[i]);
		}*/
	}

	// Impresion de resultados
	if (comprobar == OKCALC)
	{
		printf("Calculo correcto!\n");

	}

	// Impresión de resultados
	printf("Tiempo ejecución GPU : %fs\n", \
		gpu_end_time - gpu_start_time);
	printf("Tiempo de ejecución en la CPU : %fs\n", \
		cpu_end_time - cpu_start_time);
	printf("Se ha conseguido un factor de aceleración %fx utilizando CUDA\n", (cpu_end_time - cpu_start_time) / (gpu_end_time - gpu_start_time));

	// Limpieza de buffers
	BorrarSuperficie();
	if (NormalVCPU != NULL) free(NormalVCPU);
	if (NormalUCPU != NULL) free(NormalUCPU);
	if (NormalWCPU != NULL) free(NormalWCPU);
	if (NormalVGPU != NULL) free(NormalVGPU);
	if (NormalUGPU != NULL) free(NormalUGPU);
	if (NormalWGPU != NULL) free(NormalWGPU);

	hipFree(Buffer_d);
	hipFree(NormalVGPU_d);
	hipFree(NormalUGPU_d);
	hipFree(NormalWGPU_d);

	return;
}

int
main(int argc, char** argv)
{
	runTest(argc, argv);
	getchar();
}

/* Funciones auxiliares */
double getTime()
{
	timeStamp start;
	timeStamp dwFreq;
	QueryPerformanceFrequency(&dwFreq);
	QueryPerformanceCounter(&start);
	return double(start.QuadPart) / double(dwFreq.QuadPart);
}



/*----------------------------------------------------------------------------*/
/*	Función:  LeerSuperficie(char *fichero)						              */
/*													                          */
/*	          Lee los datos de la superficie de un fichero con formato .FOR   */
/*----------------------------------------------------------------------------*/
int LeerSuperficie(const char* fichero)
{
	int i, j, count;		/* Variables de bucle */
	int utotal, vtotal;		/* Variables de tamaño de superficie */
	FILE* fpin; 			/* Fichero */
	double x, y, z;

	errno_t err;

	// Open for read
	err = fopen_s(&fpin, fichero, "r");

	/* Apertura de Fichero */
	if (err != 0) return ERRORCALC;
	/* Lectura de cabecera */
	if (fscanf(fpin, "Ancho=%d\n", &utotal) < 0) return ERRORCALC;
	if (fscanf(fpin, "Alto=%d\n", &vtotal) < 0) return ERRORCALC;
	if (utotal * vtotal <= 0) return ERRORCALC;
	/* Localizacion de comienzo */
	if (feof(fpin)) return ERRORCALC;
	/* Inicialización de parametros geometricos */
	if (CrearSuperficie(utotal, vtotal) == ERRORCALC) return ERRORCALC;
	/* Lectura de coordenadas */
	count = 0;
	for (i = 0; i < utotal; i++)
	{
		for (j = 0; j < vtotal; j++)
		{
			if (!feof(fpin))
			{
				fscanf(fpin, "%lf %lf %lf\n", &x, &y, &z);
				S.Buffer[j][i].x = x;
				S.Buffer[j][i].y = y;
				S.Buffer[j][i].z = z;
				count++;
			}
			else break;
		}
	}
	fclose(fpin);
	if (count != utotal * vtotal) return ERRORCALC;
	return OKCALC;
}



